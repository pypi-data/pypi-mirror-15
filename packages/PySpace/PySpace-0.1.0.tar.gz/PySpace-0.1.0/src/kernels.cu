#include "hip/hip_runtime.h"
#include "pyspace.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#define NORM2(X, Y, Z) X*X + Y*Y + Z*Z

__device__
void calculate_force_device(double* x_old, double* y_old, double* z_old, double* m,
        double x_i, double y_i, double z_i,
        double& a_x, double& a_y, double& a_z,
        int num_planets, double eps2, double G)
{
    double r_x_j, r_y_j, r_z_j;
    double x_ji, y_ji, z_ji;
    double m_j;

    double cnst;
    double dist_ij;

    int j;
    for(j=0; j<num_planets; j++)
    {
        r_x_j = x_old[j];
        r_y_j = y_old[j];
        r_z_j = z_old[j];

        m_j = m[j];

        x_ji = r_x_j - x_i;
        y_ji = r_y_j - y_i;
        z_ji = r_z_j - z_i;

        dist_ij = sqrt(eps2 + NORM2(x_ji, y_ji, z_ji));

        if(dist_ij == 0)
            return;

        cnst = (G*m_j/(dist_ij*dist_ij*dist_ij));

        a_x += x_ji*cnst;
        a_y += y_ji*cnst;
        a_z += z_ji*cnst;
    }

}


__global__
void brute_force_kernel(double* x, double* y, double* z,
        double* x_old, double* y_old, double* z_old,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z,
        double* m, double G, double dt, int num_planets, double eps)
{
    double eps2 = eps*eps;

    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if(id >= num_planets)
        return;
    
    //Update id'th planet

    double a_x_i = a_x[id];
    double a_y_i = a_y[id];
    double a_z_i = a_z[id];

    calculate_force_device(x_old, y_old, z_old, m,
            x_old[id], y_old[id], z_old[id],
            a_x[id], a_y[id], a_z[id],
            num_planets, eps2, G);

    x[id] += v_x[id]*dt + a_x_i*0.5*dt*dt;
    y[id] += v_y[id]*dt + a_y_i*0.5*dt*dt;
    z[id] += v_z[id]*dt + a_z_i*0.5*dt*dt;

    v_x[id] += (a_x_i + a_x[id])*0.5*dt;
    v_y[id] += (a_y_i + a_y[id])*0.5*dt;
    v_z[id] += (a_z_i + a_z[id])*0.5*dt;
}


__host__
void brute_force_gpu_update(double* x, double* y, double* z,
        double* v_x, double* v_y, double* v_z,
        double* a_x, double* a_y, double* a_z,
        double* m, double G, double dt, int num_planets, double eps)
{
    double* x_old = (double*) malloc(num_planets*sizeof(double));
    double* y_old = (double*) malloc(num_planets*sizeof(double));
    double* z_old = (double*) malloc(num_planets*sizeof(double));

    int i;
    for(i=0; i<num_planets; i++)
    {
        x_old[i] = x[i];
        y_old[i] = y[i];
        z_old[i] = z[i];
    }

    double *dev_x, *dev_y, *dev_z, *dev_x_old, *dev_y_old, *dev_z_old;
    double *dev_v_x, *dev_v_y, *dev_v_z, *dev_a_x, *dev_a_y, *dev_a_z;
    double *dev_m;

    if( hipMalloc((void**)&dev_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_x_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_y_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_z_old, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_v_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_x, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_y, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_a_z, num_planets*sizeof(double)) != hipSuccess ||
        hipMalloc((void**)&dev_m, num_planets*sizeof(double)) != hipSuccess   )
    {
        fprintf(stderr, "ERROR: hipMalloc failed!");
        exit(0);
    }

    if( hipMemcpy(dev_x, x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_y, y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_z, z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_x_old, x_old, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_y_old, y_old, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_z_old, z_old, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_x, v_x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_y, v_y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_v_z, v_z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_x, a_x, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_y, a_y, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_a_z, a_z, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(dev_m, m, num_planets*sizeof(double), hipMemcpyHostToDevice) != hipSuccess )
    {
        fprintf(stderr, "ERROR: hipMemcpy from host to device failed!");
        exit(0);
    }

    brute_force_kernel<<<ceil(num_planets/1024), 1024>>>(dev_x, dev_y, dev_z,
            dev_x_old, dev_y_old, dev_z_old,
            dev_v_x, dev_v_y, dev_v_z,
            dev_a_x, dev_a_y, dev_a_z,
            dev_m, G, dt, num_planets, eps);

    if( hipMemcpy(x, dev_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(y, dev_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(z, dev_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess || 
        hipMemcpy(v_x, dev_v_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(v_y, dev_v_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(v_z, dev_v_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(a_x, dev_a_x, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(a_y, dev_a_y, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(a_z, dev_a_z, num_planets*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess )
    {
        fprintf(stderr, "ERROR: hipMemcpy from device to host failed!\n");
        hipError_t error = hipMemcpy(x, dev_x, num_planets*sizeof(double), hipMemcpyDeviceToHost);
        fprintf(stderr, hipGetErrorString(error));
        fprintf(stderr, "\n");
        exit(0);
    }

    //CUDA free

    if( hipFree(dev_x) != hipSuccess ||
        hipFree(dev_y) != hipSuccess ||
        hipFree(dev_z) != hipSuccess ||
        hipFree(dev_x_old) != hipSuccess ||
        hipFree(dev_y_old) != hipSuccess ||
        hipFree(dev_z_old) != hipSuccess ||
        hipFree(dev_v_x) != hipSuccess ||
        hipFree(dev_v_y) != hipSuccess ||
        hipFree(dev_v_z) != hipSuccess ||
        hipFree(dev_a_x) != hipSuccess ||
        hipFree(dev_a_y) != hipSuccess ||
        hipFree(dev_a_z) != hipSuccess ||
        hipFree(dev_m) != hipSuccess  )
    {
        fprintf(stderr, "ERROR: hipFree failed!");
        exit(0);
    }

    free(x_old);
    free(y_old);
    free(z_old);
}

