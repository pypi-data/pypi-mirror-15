#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>

#include "util.h"
#include "arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#define PIXELTRACE


typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_projTexture;


namespace astraCUDA {

const unsigned int g_anglesPerBlock = 16;
const unsigned int g_blockSliceSize = 32;
const unsigned int g_blockSlices = 16;

const unsigned int g_MaxAngles = 2560;

__constant__ float gC_angle_sin[g_MaxAngles];
__constant__ float gC_angle_cos[g_MaxAngles];
__constant__ float gC_angle_offset[g_MaxAngles];

static bool bindProjDataTexture(float* data, unsigned int pitch, unsigned int width, unsigned int height, hipTextureAddressMode mode = hipAddressModeBorder)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_projTexture.addressMode[0] = mode;
	gT_projTexture.addressMode[1] = mode;
	gT_projTexture.filterMode = hipFilterModeLinear;
	gT_projTexture.normalized = false;

	hipBindTexture2D(0, gT_projTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);

	// TODO: error value?

	return true;
}

__global__ void devBP(float* D_volData, unsigned int volPitch, unsigned int startAngle, bool offsets, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f ) / dims.fDetScale;
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f ) / dims.fDetScale;

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;
	const float fT_base = 0.5f*dims.iProjDets - 0.5f + 0.5f;

	if (offsets) {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];
			const float TOffset = gC_angle_offset[angle];

			const float fT = fT_base + fX * cos_theta - fY * sin_theta + TOffset;
			fVal += tex2D(gT_projTexture, fT, fA);
			fA += 1.0f;
		}

	} else {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];

			const float fT = fT_base + fX * cos_theta - fY * sin_theta;
			fVal += tex2D(gT_projTexture, fT, fA);
			fA += 1.0f;
		}

	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

// supersampling version
__global__ void devBP_SS(float* D_volData, unsigned int volPitch, unsigned int startAngle, bool offsets, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim) / dims.fDetScale;
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim) / dims.fDetScale;

	const float fSubStep = 1.0f/(dims.iRaysPerPixelDim * dims.fDetScale);

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;
	const float fT_base = 0.5f*dims.iProjDets - 0.5f + 0.5f;

	fOutputScale /= (dims.iRaysPerPixelDim * dims.iRaysPerPixelDim);

	if (offsets) {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];
			const float TOffset = gC_angle_offset[angle];

			float fT = fT_base + fX * cos_theta - fY * sin_theta + TOffset;

			for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
				float fTy = fT;
				fT += fSubStep * cos_theta;
				for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
					fVal += tex2D(gT_projTexture, fTy, fA);
					fTy -= fSubStep * sin_theta;
				}
			}
			fA += 1.0f;
		}

	} else {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];

			float fT = fT_base + fX * cos_theta - fY * sin_theta;

			for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
				float fTy = fT;
				fT += fSubStep * cos_theta;
				for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
					fVal += tex2D(gT_projTexture, fTy, fA);
					fTy -= fSubStep * sin_theta;
				}
			}
			fA += 1.0f;

		}

	}

	volData[Y*volPitch+X] += fVal * fOutputScale;
}

__global__ void devBP_SART(float* D_volData, unsigned int volPitch, float offset, float angle_sin, float angle_cos, const SDimensions dims, float fOutputScale)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f ) / dims.fDetScale;
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f ) / dims.fDetScale;

	const float fT_base = 0.5f*dims.iProjDets - 0.5f + 0.5f;

	const float fT = fT_base + fX * angle_cos - fY * angle_sin + offset;
	const float fVal = tex2D(gT_projTexture, fT, 0.5f);

	D_volData[Y*volPitch+X] += fVal * fOutputScale;
}


bool BP_internal(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const float* angles, const float* TOffsets, float fOutputScale)
{
	// TODO: process angles block by block
	assert(dims.iProjAngles <= g_MaxAngles);

	float* angle_sin = new float[dims.iProjAngles];
	float* angle_cos = new float[dims.iProjAngles];

	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		angle_sin[i] = sinf(angles[i]);
		angle_cos[i] = cosf(angles[i]);
	}
	hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_sin), angle_sin, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e2 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_cos), angle_cos, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	assert(e1 == hipSuccess);
	assert(e2 == hipSuccess);

	if (TOffsets) {
		hipError_t e3 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_offset), TOffsets, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
		assert(e3 == hipSuccess);
	}

	delete[] angle_sin;
	delete[] angle_cos;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {

		if (dims.iRaysPerPixelDim > 1)
			devBP_SS<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, (TOffsets != 0), dims, fOutputScale);
		else
			devBP<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, (TOffsets != 0), dims, fOutputScale);
	}
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipStreamDestroy(stream);

	return true;
}

bool BP(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const float* angles, const float* TOffsets, float fOutputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = BP_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle,
		                  TOffsets ? TOffsets + iAngle : 0,
		                  fOutputScale);
		if (!ret)
			return false;
	}
	return true;
}


bool BP_SART(float* D_volumeData, unsigned int volumePitch,
             float* D_projData, unsigned int projPitch,
             unsigned int angle, const SDimensions& dims,
             const float* angles, const float* TOffsets, float fOutputScale)
{
	// Only one angle.
	// We need to Clamp to the border pixels instead of to zero, because
	// SART weights with ray length.
	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, 1, hipAddressModeClamp);

	float angle_sin = sinf(angles[angle]);
	float angle_cos = cosf(angles[angle]);

	float offset = 0.0f;
	if (TOffsets)
		offset = TOffsets[angle];

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	devBP_SART<<<dimGrid, dimBlock>>>(D_volumeData, volumePitch, offset, angle_sin, angle_cos, dims, fOutputScale);
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	return true;
}


}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_projData;

	SDimensions dims;
	dims.iVolWidth = 1024;
	dims.iVolHeight = 1024;
	dims.iProjAngles = 512;
	dims.iProjDets = 1536;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;

	unsigned int volumePitch, projPitch;

	allocateVolume(D_volumeData, dims.iVolWidth, dims.iVolHeight, volumePitch);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_projData, dims.iProjDets, dims.iProjAngles, projPitch);
	printf("pitch: %u\n", projPitch);

	unsigned int y, x;
	float* sino = loadImage("sino.png", y, x);

	float* img = new float[dims.iVolWidth*dims.iVolHeight];

	memset(img, 0, dims.iVolWidth*dims.iVolHeight*sizeof(float));

	copyVolumeToDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);
	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_projData, projPitch);

	float* angle = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i)
		angle[i] = i*(M_PI/dims.iProjAngles);

	BP(D_volumeData, volumePitch, D_projData, projPitch, dims, angle, 0, 1.0f);

	delete[] angle;

	copyVolumeFromDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);

	saveImage("vol.png",dims.iVolHeight,dims.iVolWidth,img);

	return 0;
}
#endif
