/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cassert>

#include "algo.h"
#include "par_fp.h"
#include "fan_fp.h"
#include "par_bp.h"
#include "fan_bp.h"
#include "util.h"
#include "arith.h"

namespace astraCUDA {

ReconAlgo::ReconAlgo()
{
	angles = 0;
	TOffsets = 0;
	fanProjs = 0;
	shouldAbort = false;

	useVolumeMask = false;
	useSinogramMask = false;
	D_maskData = 0;
	D_smaskData = 0;

	D_sinoData = 0;
	D_volumeData = 0;

	useMinConstraint = false;
	useMaxConstraint = false;

	freeGPUMemory = false;
}

ReconAlgo::~ReconAlgo()
{
	reset();
}

void ReconAlgo::reset()
{
	delete[] angles;
	delete[] TOffsets;
	delete[] fanProjs;

	if (freeGPUMemory) {
		hipFree(D_maskData);
		hipFree(D_smaskData);
		hipFree(D_sinoData);
		hipFree(D_volumeData);
	}

	angles = 0;
	TOffsets = 0;
	fanProjs = 0;
	shouldAbort = false;

	useVolumeMask = false;
	useSinogramMask = false;

	D_maskData = 0;
	D_smaskData = 0;

	D_sinoData = 0;
	D_volumeData = 0;
	
	useMinConstraint = false;
	useMaxConstraint = false;

	freeGPUMemory = false;
}

bool ReconAlgo::setGPUIndex(int iGPUIndex)
{
	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	return true;
}

bool ReconAlgo::enableVolumeMask()
{
	useVolumeMask = true;
	return true;
}

bool ReconAlgo::enableSinogramMask()
{
	useSinogramMask = true;
	return true;
}


bool ReconAlgo::setGeometry(const SDimensions& _dims, const float* _angles)
{
	dims = _dims;

	angles = new float[dims.iProjAngles];

	memcpy(angles, _angles, sizeof(angles[0]) * dims.iProjAngles);

	delete[] fanProjs;
	fanProjs = 0;

	return true;
}

bool ReconAlgo::setFanGeometry(const SDimensions& _dims,
                               const SFanProjection* _projs)
{
	dims = _dims;
	fanProjs = new SFanProjection[dims.iProjAngles];

	memcpy(fanProjs, _projs, sizeof(fanProjs[0]) * dims.iProjAngles);

	delete[] angles;
	angles = 0;

	return true;
}


bool ReconAlgo::setTOffsets(const float* _TOffsets)
{
	// TODO: determine if they're all zero?
	TOffsets = new float[dims.iProjAngles];
	memcpy(TOffsets, _TOffsets, sizeof(angles[0]) * dims.iProjAngles);

	return true;
}



bool ReconAlgo::setVolumeMask(float* _D_maskData, unsigned int _maskPitch)
{
	assert(useVolumeMask);

	D_maskData = _D_maskData;
	maskPitch = _maskPitch;

	return true;
}

bool ReconAlgo::setSinogramMask(float* _D_smaskData, unsigned int _smaskPitch)
{
	assert(useSinogramMask);

	D_smaskData = _D_smaskData;
	smaskPitch = _smaskPitch;

	return true;
}

bool ReconAlgo::setBuffers(float* _D_volumeData, unsigned int _volumePitch,
                      float* _D_projData, unsigned int _projPitch)
{
	D_volumeData = _D_volumeData;
	volumePitch = _volumePitch;
	D_sinoData = _D_projData;
	sinoPitch = _projPitch;

	return true;
}

bool ReconAlgo::setMinConstraint(float fMin)
{
	fMinConstraint = fMin;
	useMinConstraint = true;
	return true;
}

bool ReconAlgo::setMaxConstraint(float fMax)
{
	fMaxConstraint = fMax;
	useMaxConstraint = true;
	return true;
}



bool ReconAlgo::allocateBuffers()
{
	bool ok;
	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		D_volumeData = 0;
		return false;
	}

	if (useVolumeMask) {
		ok = allocateVolumeData(D_maskData, maskPitch, dims);
		if (!ok) {
			hipFree(D_volumeData);
			hipFree(D_sinoData);
			D_volumeData = 0;
			D_sinoData = 0;
			return false;
		}
	}

	if (useSinogramMask) {
		ok = allocateProjectionData(D_smaskData, smaskPitch, dims);
		if (!ok) {
			hipFree(D_volumeData);
			hipFree(D_sinoData);
			hipFree(D_maskData);
			D_volumeData = 0;
			D_sinoData = 0;
			D_maskData = 0;
			return false;
		}
	}

	freeGPUMemory = true;
	return true;
}

bool ReconAlgo::copyDataToGPU(const float* pfSinogram, unsigned int iSinogramPitch, float fSinogramScale,
                              const float* pfReconstruction, unsigned int iReconstructionPitch,
                              const float* pfVolMask, unsigned int iVolMaskPitch,
                              const float* pfSinoMask, unsigned int iSinoMaskPitch)
{
	if (!pfSinogram)
		return false;
	if (!pfReconstruction)
		return false;

	bool ok = copySinogramToDevice(pfSinogram, iSinogramPitch,
	                               dims,
	                               D_sinoData, sinoPitch);
	if (!ok)
		return false;

	// rescale sinogram to adjust for pixel size
	processSino<opMul>(D_sinoData, fSinogramScale,
	                       //1.0f/(fPixelSize*fPixelSize),
	                       sinoPitch, dims);

	ok = copyVolumeToDevice(pfReconstruction, iReconstructionPitch,
	                        dims,
	                        D_volumeData, volumePitch);
	if (!ok)
		return false;



	if (useVolumeMask) {
		if (!pfVolMask)
			return false;

		ok = copyVolumeToDevice(pfVolMask, iVolMaskPitch,
		                        dims,
		                        D_maskData, maskPitch);
		if (!ok)
			return false;
	}

	if (useSinogramMask) {
		if (!pfSinoMask)
			return false;

		ok = copySinogramToDevice(pfSinoMask, iSinoMaskPitch,
		                          dims,
		                          D_smaskData, smaskPitch);
		if (!ok)
			return false;
	}

	return true;
}

bool ReconAlgo::getReconstruction(float* pfReconstruction,
                                  unsigned int iReconstructionPitch) const
{
	bool ok = copyVolumeFromDevice(pfReconstruction, iReconstructionPitch,
	                               dims,
	                               D_volumeData, volumePitch);
	if (!ok)
		return false;

	return true;
}


bool ReconAlgo::callFP(float* D_volumeData, unsigned int volumePitch,
                       float* D_projData, unsigned int projPitch,
                       float outputScale)
{
	if (angles) {
		assert(!fanProjs);
		return FP(D_volumeData, volumePitch, D_projData, projPitch,
		          dims, angles, TOffsets, outputScale);
	} else {
		assert(fanProjs);
		return FanFP(D_volumeData, volumePitch, D_projData, projPitch,
		             dims, fanProjs, outputScale);
	}
}

bool ReconAlgo::callBP(float* D_volumeData, unsigned int volumePitch,
                       float* D_projData, unsigned int projPitch,
                       float outputScale)
{
	if (angles) {
		assert(!fanProjs);
		return BP(D_volumeData, volumePitch, D_projData, projPitch,
		          dims, angles, TOffsets, outputScale);
	} else {
		assert(fanProjs);
		return FanBP(D_volumeData, volumePitch, D_projData, projPitch,
		             dims, fanProjs, outputScale);
	}

}



}
