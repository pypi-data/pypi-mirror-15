#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2015, iMinds-Vision Lab, University of Antwerp
           2014-2015, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "cone_fp.h"
#include "testutil.h"
#endif

#include "dims3d.h"
#include "arith3d.h"
#include "../2d/fft.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneProjTexture;

namespace astraCUDA3d {

static const unsigned int g_volBlockZ = 16;

static const unsigned int g_anglesPerBlock = 64;
static const unsigned int g_volBlockX = 32;
static const unsigned int g_volBlockY = 16;

static const unsigned int g_anglesPerWeightBlock = 16;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 2048;

__constant__ float gC_angle_sin[g_MaxAngles];
__constant__ float gC_angle_cos[g_MaxAngles];
__constant__ float gC_angle[g_MaxAngles];


// per-detector u/v shifts?

static bool bindProjDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneProjTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneProjTexture.filterMode = hipFilterModeLinear;
	gT_coneProjTexture.normalized = false;

	hipBindTextureToArray(gT_coneProjTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


__global__ void devBP_FDK(void* D_volData, unsigned int volPitch, int startAngle, float fSrcOrigin, float fDetOrigin, float fSrcZ, float fDetZ, float fInvDetUSize, float fInvDetVSize, const SDimensions3D dims)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X > dims.iVolX)
		return;
	if (Y > dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f;
	float fY = Y - 0.5f*dims.iVolY + 0.5f;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - fSrcZ;

	const float fU_base = 0.5f*dims.iProjU - 0.5f + 0.5f;
	const float fV_base = 0.5f*dims.iProjV - 0.5f + 0.5f + (fDetZ-fSrcZ);

	// Note re. fZ/rV_base: the computations below are all relative to the
	// optical axis, so we do the Z-adjustments beforehand.

	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{

			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];

			const float fR = fSrcOrigin;
			const float fD = fR - fX * sin_theta + fY * cos_theta;
			float fWeight = fR / fD;
			fWeight *= fWeight;

			const float fScaleFactor = (fR + fDetOrigin) / fD;
			const float fU = fU_base + (fX*cos_theta+fY*sin_theta) * fScaleFactor * fInvDetUSize;
			const float fV = fV_base + fZ * fScaleFactor * fInvDetVSize;

			fVal += tex3D(gT_coneProjTexture, fU, fAngle, fV);

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal;
//		projData[(angle*dims.iProjV+detectorV)*projPitch+detectorU] = 10.0f;
//		if (threadIdx.x == 0 && threadIdx.y == 0) { printf("%d,%d,%d [%d / %d] -> %f\n", angle, detectorU, detectorV, (angle*dims.iProjV+detectorV)*projPitch+detectorU, projPitch, projData[(angle*dims.iProjV+detectorV)*projPitch+detectorU]); }
	}

}


bool FDK_BP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            float fSrcOrigin, float fDetOrigin,
            float fSrcZ, float fDetZ, float fDetUSize, float fDetVSize,
            const SDimensions3D& dims, const float* angles)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bindProjDataTexture(cuArray);

	float* angle_sin = new float[dims.iProjAngles];
	float* angle_cos = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		angle_sin[i] = sinf(angles[i]);
		angle_cos[i] = cosf(angles[i]);
	}
	hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_sin), angle_sin, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e2 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_cos), angle_cos, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	assert(e1 == hipSuccess);
	assert(e2 == hipSuccess);

	delete[] angle_sin;
	delete[] angle_cos;

	dim3 dimBlock(g_volBlockX, g_volBlockY);

	dim3 dimGrid(((dims.iVolX+g_volBlockX-1)/g_volBlockX)*((dims.iVolY+g_volBlockY-1)/g_volBlockY), (dims.iVolZ+g_volBlockZ-1)/g_volBlockZ);

	// timeval t;
	// tic(t);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {
		devBP_FDK<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, fSrcOrigin, fDetOrigin, fSrcZ, fDetZ, 1.0f / fDetUSize, 1.0f / fDetVSize, dims);
	}

	cudaTextForceKernelsCompletion();

	hipFreeArray(cuArray);

	// printf("%f\n", toc(t));

	return true;
}

__global__ void devFDK_preweight(void* D_projData, unsigned int projPitch, unsigned int startAngle, unsigned int endAngle, float fSrcOrigin, float fDetOrigin, float fSrcZ, float fDetZ, float fDetUSize, float fDetVSize, const SDimensions3D dims)
{
	float* projData = (float*)D_projData;
	int angle = startAngle + blockIdx.y * g_anglesPerWeightBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	// We need the length of the central ray and the length of the ray(s) to
	// our detector pixel(s).

	const float fCentralRayLength = fSrcOrigin + fDetOrigin;

	const float fU = (detectorU - 0.5f*dims.iProjU + 0.5f) * fDetUSize;

	const float fT = fCentralRayLength * fCentralRayLength + fU * fU;

	float fV = (startDetectorV - 0.5f*dims.iProjV + 0.5f) * fDetVSize + fDetZ - fSrcZ;

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{
		const float fRayLength = sqrtf(fT + fV * fV);

		const float fWeight = fCentralRayLength / fRayLength;

		projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] *= fWeight;

		fV += 1.0f;
	}
}

__global__ void devFDK_ParkerWeight(void* D_projData, unsigned int projPitch, unsigned int startAngle, unsigned int endAngle, float fSrcOrigin, float fDetOrigin, float fSrcZ, float fDetZ, float fDetUSize, float fCentralFanAngle, const SDimensions3D dims)
{
	float* projData = (float*)D_projData;
	int angle = startAngle + blockIdx.y * g_anglesPerWeightBlock + threadIdx.y;
	if (angle >= endAngle)
		return;

	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;
	int endDetectorV = startDetectorV + g_detBlockV;
	if (endDetectorV > dims.iProjV)
		endDetectorV = dims.iProjV;

	// We need the length of the central ray and the length of the projection
	// of our ray onto the central slice

	const float fCentralRayLength = fSrcOrigin + fDetOrigin;

	// TODO: Detector pixel size
	const float fU = (detectorU - 0.5f*dims.iProjU + 0.5f) * fDetUSize;

	//const float fGamma = atanf(fU / fCentralRayLength);
	//const float fBeta = gC_angle[angle];
	const float fGamma = atanf(fU / fCentralRayLength);
	float fBeta = -gC_angle[angle];
	if (fBeta < 0.0f)
		fBeta += 2*M_PI;
	if (fBeta >= 2*M_PI)
		fBeta -= 2*M_PI;

	// compute the weight depending on the location in the central fan's radon
	// space
	float fWeight;

	if (fBeta <= 0.0f) {
		fWeight = 0.0f;
	} else if (fBeta <= 2.0f*(fCentralFanAngle + fGamma)) {
		fWeight = sinf((M_PI / 4.0f) * fBeta / (fCentralFanAngle + fGamma));
		fWeight *= fWeight;
	} else if (fBeta <= M_PI + 2*fGamma) {
		fWeight = 1.0f;
	} else if (fBeta <= M_PI + 2*fCentralFanAngle) {
		fWeight = sinf((M_PI / 4.0f) * (M_PI + 2.0f*fCentralFanAngle - fBeta) / (fCentralFanAngle - fGamma));
		fWeight *= fWeight;
	} else {
		fWeight = 0.0f;
	}

	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)
	{

		projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] *= fWeight;

	}
}



// Perform the FDK pre-weighting and filtering
bool FDK_PreWeight(hipPitchedPtr D_projData,
                float fSrcOrigin, float fDetOrigin,
                float fSrcZ, float fDetZ,
                float fDetUSize, float fDetVSize, bool bShortScan,
                const SDimensions3D& dims, const float* angles)
{
	// The pre-weighting factor for a ray is the cosine of the angle between
	// the central line and the ray.

	dim3 dimBlock(g_detBlockU, g_anglesPerWeightBlock);
	dim3 dimGrid( ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
	              (dims.iProjAngles+g_anglesPerWeightBlock-1)/g_anglesPerWeightBlock);

	int projPitch = D_projData.pitch/sizeof(float);

	devFDK_preweight<<<dimGrid, dimBlock>>>(D_projData.ptr, projPitch, 0, dims.iProjAngles, fSrcOrigin, fDetOrigin, fSrcZ, fDetZ, fDetUSize, fDetVSize, dims);

	cudaTextForceKernelsCompletion();

	if (bShortScan) {
		// We do short-scan Parker weighting

		hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle), angles,
		                                    dims.iProjAngles*sizeof(float), 0,
		                                    hipMemcpyHostToDevice);
		assert(!e1);

		// TODO: detector pixel size!
		float fCentralFanAngle = atanf((dims.iProjU*0.5f) /
		                               (fSrcOrigin + fDetOrigin));

		devFDK_ParkerWeight<<<dimGrid, dimBlock>>>(D_projData.ptr, projPitch, 0, dims.iProjAngles, fSrcOrigin, fDetOrigin, fSrcZ, fDetZ, fDetUSize, fCentralFanAngle, dims);

	}

	cudaTextForceKernelsCompletion();
	return true;
}

bool FDK_Filter(hipPitchedPtr D_projData,
                hipfftComplex * D_filter,
                float fSrcOrigin, float fDetOrigin,
                float fSrcZ, float fDetZ,
                float fDetUSize, float fDetVSize, bool bShortScan,
                const SDimensions3D& dims, const float* angles)
{

	// The filtering is a regular ramp filter per detector line.

	int iPaddedDetCount = calcNextPowerOfTwo(2 * dims.iProjU);
	int iHalfFFTSize = calcFFTFourSize(iPaddedDetCount);
	int projPitch = D_projData.pitch/sizeof(float);
	

	// We process one sinogram at a time.
	float* D_sinoData = (float*)D_projData.ptr;

	hipfftComplex * D_sinoFFT = NULL;
	allocateComplexOnDevice(dims.iProjAngles, iHalfFFTSize, &D_sinoFFT);

	bool ok = true;

	for (int v = 0; v < dims.iProjV; ++v) {

		ok = runCudaFFT(dims.iProjAngles, D_sinoData, projPitch,
		                dims.iProjU, iPaddedDetCount, iHalfFFTSize,
		                D_sinoFFT);

		if (!ok) break;

		applyFilter(dims.iProjAngles, iHalfFFTSize, D_sinoFFT, D_filter);


		ok = runCudaIFFT(dims.iProjAngles, D_sinoFFT, D_sinoData, projPitch,
		                 dims.iProjU, iPaddedDetCount, iHalfFFTSize);

		if (!ok) break;

		D_sinoData += (dims.iProjAngles * projPitch);
	}

	freeComplexOnDevice(D_sinoFFT);

	return ok;
}


bool FDK(hipPitchedPtr D_volumeData,
         hipPitchedPtr D_projData,
         float fSrcOrigin, float fDetOrigin,
         float fSrcZ, float fDetZ, float fDetUSize, float fDetVSize,
         const SDimensions3D& dims, const float* angles, bool bShortScan)
{
	bool ok;
	// Generate filter
	// TODO: Check errors
	hipfftComplex * D_filter;
	int iPaddedDetCount = calcNextPowerOfTwo(2 * dims.iProjU);
	int iHalfFFTSize = calcFFTFourSize(iPaddedDetCount);

	ok = FDK_PreWeight(D_projData, fSrcOrigin, fDetOrigin,
	                fSrcZ, fDetZ, fDetUSize, fDetVSize,
	                bShortScan, dims, angles);
	if (!ok)
		return false;

	hipfftComplex *pHostFilter = new hipfftComplex[dims.iProjAngles * iHalfFFTSize];
	memset(pHostFilter, 0, sizeof(hipfftComplex) * dims.iProjAngles * iHalfFFTSize);

	genFilter(FILTER_RAMLAK, 1.0f, dims.iProjAngles, pHostFilter, iPaddedDetCount, iHalfFFTSize);


	allocateComplexOnDevice(dims.iProjAngles, iHalfFFTSize, &D_filter);
	uploadComplexArrayToDevice(dims.iProjAngles, iHalfFFTSize, pHostFilter, D_filter);

	delete [] pHostFilter;


	// Perform filtering



	ok = FDK_Filter(D_projData, D_filter, fSrcOrigin, fDetOrigin,
	                fSrcZ, fDetZ, fDetUSize, fDetVSize,
	                bShortScan, dims, angles);

	// Clean up filter
	freeComplexOnDevice(D_filter);


	if (!ok)
		return false;

	// Perform BP

	ok = FDK_BP(D_volumeData, D_projData, fSrcOrigin, fDetOrigin, fSrcZ, fDetZ,
	            fDetUSize, fDetVSize, dims, angles);

	if (!ok)
		return false;

	processVol3D<opMul>(D_volumeData,
	                  (M_PI / 2.0f) / (float)dims.iProjAngles, dims);

	return true;
}


}

#ifdef STANDALONE
void dumpVolume(const char* filespec, const hipPitchedPtr& data, const SDimensions3D& dims, float fMin, float fMax)
{
	float* buf = new float[dims.iVolX*dims.iVolY];
	unsigned int pitch = data.pitch / sizeof(float);

	for (int i = 0; i < dims.iVolZ; ++i) {
		hipMemcpy2D(buf, dims.iVolX*sizeof(float), ((float*)data.ptr)+pitch*dims.iVolY*i, data.pitch, dims.iVolX*sizeof(float), dims.iVolY, hipMemcpyDeviceToHost);

		char fname[512];
		sprintf(fname, filespec, dims.iVolZ-i-1);
		saveImage(fname, dims.iVolY, dims.iVolX, buf, fMin, fMax);
	}
}

void dumpSinograms(const char* filespec, const hipPitchedPtr& data, const SDimensions3D& dims, float fMin, float fMax)
{
	float* bufs = new float[dims.iProjAngles*dims.iProjU];
	unsigned int pitch = data.pitch / sizeof(float);

	for (int i = 0; i < dims.iProjV; ++i) {
		hipMemcpy2D(bufs, dims.iProjU*sizeof(float), ((float*)data.ptr)+pitch*dims.iProjAngles*i, data.pitch, dims.iProjU*sizeof(float), dims.iProjAngles, hipMemcpyDeviceToHost);

		char fname[512];
		sprintf(fname, filespec, i);
		saveImage(fname, dims.iProjAngles, dims.iProjU, bufs, fMin, fMax);
	}
}

void dumpProjections(const char* filespec, const hipPitchedPtr& data, const SDimensions3D& dims, float fMin, float fMax)
{
	float* bufp = new float[dims.iProjV*dims.iProjU];
	unsigned int pitch = data.pitch / sizeof(float);

	for (int i = 0; i < dims.iProjAngles; ++i) {
		for (int j = 0; j < dims.iProjV; ++j) {
			hipMemcpy(bufp+dims.iProjU*j, ((float*)data.ptr)+pitch*dims.iProjAngles*j+pitch*i, dims.iProjU*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[512];
		sprintf(fname, filespec, i);
		saveImage(fname, dims.iProjV, dims.iProjU, bufp, fMin, fMax);
	}
}




int main()
{
#if 0
	SDimensions3D dims;
	dims.iVolX = 512;
	dims.iVolY = 512;
	dims.iVolZ = 512;
	dims.iProjAngles = 180;
	dims.iProjU = 1024;
	dims.iProjV = 1024;
	dims.iRaysPerDet = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

#if 0
	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
#if 0
		if (i == 128) {
			for (unsigned int j = 0; j < 256*256; ++j)
				slice[j] = 0.0f;
		}
#endif 
	}
#endif

	SConeProjection angle[180];
	angle[0].fSrcX = -1536;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 0;

	angle[0].fDetSX = 1024;
	angle[0].fDetSY = -512;
	angle[0].fDetSZ = 512;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = -1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 180; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*2*M_PI/180);
		ROTATE0(DetS, i, i*2*M_PI/180);
		ROTATE0(DetU, i, i*2*M_PI/180);
		ROTATE0(DetV, i, i*2*M_PI/180);
	}
#undef ROTATE0

	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);

	//dumpSinograms("sino%03d.png", projData, dims, 0, 512);
	//dumpProjections("proj%03d.png", projData, dims, 0, 512);

	astraCUDA3d::zeroVolumeData(volData, dims);

	float* angles = new float[dims.iProjAngles];
	for (int i = 0; i < 180; ++i)
		angles[i] = i*2*M_PI/180;

	astraCUDA3d::FDK(volData, projData, 1536, 512, 0, 0, dims, angles);

	dumpVolume("vol%03d.png", volData, dims, -20, 100);


#else

	SDimensions3D dims;
	dims.iVolX = 1000;
	dims.iVolY = 999;
	dims.iVolZ = 500;
	dims.iProjAngles = 376;
	dims.iProjU = 1024;
	dims.iProjV = 524;
	dims.iRaysPerDet = 1;

	float* angles = new float[dims.iProjAngles];
	for (int i = 0; i < dims.iProjAngles; ++i)
		angles[i] = -i*(M_PI)/360;

	hipPitchedPtr volData = astraCUDA3d::allocateVolumeData(dims);
	hipPitchedPtr projData = astraCUDA3d::allocateProjectionData(dims);
	astraCUDA3d::zeroProjectionData(projData, dims);
	astraCUDA3d::zeroVolumeData(volData, dims);

	timeval t;
	tic(t);

	for (int i = 0; i < dims.iProjAngles; ++i) {
		char fname[256];
		sprintf(fname, "/home/wpalenst/tmp/Elke/proj%04d.png", i);
		unsigned int w,h;
		float* bufp = loadImage(fname, w,h);

		int pitch = projData.pitch / sizeof(float);
		for (int j = 0; j < dims.iProjV; ++j) {
			hipMemcpy(((float*)projData.ptr)+dims.iProjAngles*pitch*j+pitch*i, bufp+dims.iProjU*j, dims.iProjU*sizeof(float), hipMemcpyHostToDevice);
		}

		delete[] bufp;
	}
	printf("Load time: %f\n", toc(t));

	//dumpSinograms("sino%03d.png", projData, dims, -8.0f, 256.0f);
	//astraCUDA3d::FDK(volData, projData, 7350, 62355, 0, 10, dims, angles);
	//astraCUDA3d::FDK(volData, projData, 7350, -380, 0, 10, dims, angles);

	tic(t);

	astraCUDA3d::FDK(volData, projData, 7383.29867, 0, 0, 10, dims, angles);

	printf("FDK time: %f\n", toc(t));
	tic(t);

	dumpVolume("vol%03d.png", volData, dims, -65.9f, 200.0f);
	//dumpVolume("vol%03d.png", volData, dims, 0.0f, 256.0f);
	printf("Save time: %f\n", toc(t));

#endif


}
#endif
